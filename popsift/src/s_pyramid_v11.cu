#include "hip/hip_runtime.h"
#include "s_pyramid.h"

#include "write_plane_2d.h"
#include "gauss_filter.h"
#include "clamp.h"
#include "debug_macros.h"
#include "assist.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <>

/*************************************************************
 * V11: device side
 *************************************************************/

#define V11_EDGE_LEN 32
#define V11_RANGE    4 // RANGES from 1 to 8 are possible

namespace popart {

__global__
void filter_gauss_horiz_v11( Plane2D_float src_data,
                             Plane2D_float dst_data )
{
    __shared__ float loaddata[V11_EDGE_LEN][V11_RANGE + V11_EDGE_LEN + V11_RANGE];

    const int src_w = src_data.getWidth();
    const int src_h = src_data.getHeight();

    int idx     = threadIdx.x;
    int idy     = threadIdx.y;
    for( ; idx < V11_EDGE_LEN+2*V11_RANGE; idx += V11_EDGE_LEN) {
        int read_x = clamp( blockIdx.x * blockDim.x + idx - V11_RANGE, src_w );
        int read_y = clamp( blockIdx.y * blockDim.y + idy,             src_h );
        loaddata[idy][idx] = src_data.ptr(read_y)[read_x];
    }
    __syncthreads();

    float g;
    float val;
    float out = 0;

    for( int offset = V11_RANGE; offset>0; offset-- ) {
        g  = popart::d_gauss_filter[GAUSS_ONE_SIDE_RANGE - offset];
        val = loaddata[threadIdx.y][threadIdx.x+V11_RANGE-offset];
        out += ( val * g );
        val = loaddata[threadIdx.y][threadIdx.x+V11_RANGE+offset];
        out += ( val * g );
    }
    g  = popart::d_gauss_filter[GAUSS_ONE_SIDE_RANGE];
    val = loaddata[threadIdx.y][threadIdx.x+V11_RANGE];
    out += ( val * g );

    idx = blockIdx.x * blockDim.x + threadIdx.x;
    idy = blockIdx.y * blockDim.y + threadIdx.y;
    if( idx >= src_w ) return;
    if( idy >= src_h ) return;

    dst_data.ptr(idy)[idx] = out;
}

__global__
void filter_gauss_horiz_v11( hipTextureObject_t src_data,
                             Plane2D_float       dst_data )
{
    int block_x = blockIdx.x * blockDim.x;
    int block_y = blockIdx.y * blockDim.y;
    int idx;
    int idy     = threadIdx.y;

    float g;
    float val;
    float out = 0;

    for( int offset = V11_RANGE; offset>0; offset-- ) {
        g  = popart::d_gauss_filter[GAUSS_ONE_SIDE_RANGE - offset];

        idx = threadIdx.x - offset;
        val = tex2D<float>( src_data, block_x + idx, block_y + idy );
        out += ( val * g );

        idx = threadIdx.x + offset;
        val = tex2D<float>( src_data, block_x + idx, block_y + idy );
        out += ( val * g );
    }

    g  = popart::d_gauss_filter[GAUSS_ONE_SIDE_RANGE];
    idx = threadIdx.x;
    val = tex2D<float>( src_data, block_x + idx, block_y + idy );
    out += ( val * g );

    idx = block_x+threadIdx.x;
    idy = block_y+threadIdx.y;
    const int dst_w = dst_data.getWidth();
    const int dst_h = dst_data.getHeight();
    if( idx >= dst_w ) return;
    if( idy >= dst_h ) return;

    dst_data.ptr(idy)[idx] = out;
}

__global__
void filter_gauss_horiz_v11_by_2( hipTextureObject_t src_data,
                                  Plane2D_float       dst_data )
{
    int block_x = blockIdx.x * blockDim.x;
    int block_y = blockIdx.y * blockDim.y;
    int idx;
    int idy     = threadIdx.y;

    float g;
    float val;
    float out = 0;

    for( int offset = V11_RANGE; offset>0; offset-- ) {
        g  = popart::d_gauss_filter[GAUSS_ONE_SIDE_RANGE - offset];

        idx = threadIdx.x - offset;
        val = tex2D<float>( src_data, 2 * ( block_x + idx ), 2 * ( block_y + idy ) );
        out += ( val * g );

        idx = threadIdx.x + offset;
        val = tex2D<float>( src_data, 2 * ( block_x + idx ), 2 * ( block_y + idy ) );
        out += ( val * g );
    }

    g  = popart::d_gauss_filter[GAUSS_ONE_SIDE_RANGE];
    idx = threadIdx.x;
    val = tex2D<float>( src_data, 2 * ( block_x + idx ), 2 * ( block_y + idy ) );
    out += ( val * g );

    idx = block_x+threadIdx.x;
    idy = block_y+threadIdx.y;
    const int dst_w = dst_data.getWidth();
    const int dst_h = dst_data.getHeight();
    if( idx >= dst_w ) return;
    if( idy >= dst_h ) return;

    dst_data.ptr(idy)[idx] = out;
}

    //input texture (src_data) has twize the size of dst_data.
    //the block and thread dimensions are that of dst_data.
__global__
void downscale_by_2(Plane2D_float src_data,
                    Plane2D_float dst_data)
{
    int block_x = blockIdx.x * blockDim.x;
    int block_y = blockIdx.y * blockDim.y;
    int idx     = threadIdx.x;
    int idy     = threadIdx.y;

    const int dst_w = dst_data.getWidth();
    const int dst_h = dst_data.getHeight();
    if( idx >= dst_w ) return;
    if( idy >= dst_h ) return;

    //todo: cant do tex2d lookup in Plane2D_float array (not texture memory).
    //      Need to either use another input buffer, or change to slower global memory lookup.
    //add 0.5f to lookup coords to get interpolated values? Does it work here?
    dst_data.ptr(idy)[idx] = tex2D<float>( src_data,
                                           2 * ( block_x + idx ) /*+ 0.5f*/,
                                           2 * ( block_y + idy )/* + 0.5f */);
    */
}

__device__ inline
float filter_gauss_vert_v11_sub( hipTextureObject_t src_data,
                                 Plane2D_float       dst_data )
{
    int block_x = blockIdx.x * blockDim.x;
    int block_y = blockIdx.y * blockDim.y;
    int idx     = threadIdx.x;
    int idy;

    float g;
    float val;
    float out = 0;

    for( int offset = V11_RANGE; offset>0; offset-- ) {
        g  = popart::d_gauss_filter[GAUSS_ONE_SIDE_RANGE - offset];

        idy = threadIdx.y - offset;
        val = tex2D<float>( src_data, block_x + idx, block_y + idy );
        out += ( val * g );

        idy = threadIdx.y + offset;
        val = tex2D<float>( src_data, block_x + idx, block_y + idy );
        out += ( val * g );
    }

    g  = popart::d_gauss_filter[GAUSS_ONE_SIDE_RANGE];
    idy = threadIdx.y;
    val = tex2D<float>( src_data, block_x + idx, block_y + idy );
    out += ( val * g );

    idx = block_x+threadIdx.x;
    idy = block_y+threadIdx.y;
    const int dst_w = dst_data.getWidth();
    const int dst_h = dst_data.getHeight();
    if( idx >= dst_w ) return 0;
    if( idy >= dst_h ) return 0;

    dst_data.ptr(idy)[idx] = out;

    return out;
}

__global__
void filter_gauss_vert_v11( hipTextureObject_t src_data,
                            Plane2D_float       dst_data )
{
    filter_gauss_vert_v11_sub( src_data, dst_data );
}

__global__
void filter_gauss_vert_v11_dog( hipTextureObject_t src_data,
                                Plane2D_float       dst_data,
                                hipTextureObject_t top_data,
                                hipSurfaceObject_t dog_data,
                                int                 level )
{
    float b = filter_gauss_vert_v11_sub( src_data, dst_data );

    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int idy = blockIdx.y * blockDim.y + threadIdx.y;

    float a;
    a = tex2D<float>( top_data, idx, idy );
    a = fabs( a - b );

    surf2DLayeredwrite( a, dog_data,
                        idx*4, idy, level,
                        hipBoundaryModeZero );
}

/*************************************************************
 * V11: host side
 *************************************************************/
__host__
void Pyramid::build_v11( Image* base )
{
#if (PYRAMID_PRINT_DEBUG==1)
    cerr << "Entering " << __FUNCTION__ << " with base image "  << endl
         << "    type size         : " << base->type_size << endl
         << "    aligned byte size : " << base->a_width << "x" << base->a_height << endl
         << "    pitch size        : " << base->pitch << "x" << base->a_height << endl
         << "    original byte size: " << base->u_width << "x" << base->u_height << endl
         << "    aligned pix size  : " << base->a_width/base->type_size << "x" << base->a_height << endl
         << "    original pix size : " << base->u_width/base->type_size << "x" << base->u_height << endl;
#endif // (PYRAMID_PRINT_DEBUG==1)

#if 0
    //Creating the octaves
    for(uint32_t octave=0; octave<_num_octaves; octave++){
        const int width  = _octaves[octave].getData(0).getWidth();
        const int height = _octaves[octave].getData(0).getHeight();
        dim3 h_block( 64, 2 );
        dim3 h_grid;

        h_grid.x = (unsigned int)grid_divide( width,  h_block.x );
        h_grid.y = (unsigned int)grid_divide( height, h_block.y );

        dim3 v_block( 64, 2 );
        dim3 v_grid;
        v_grid.x = (unsigned int)grid_divide( width,  v_block.x );
        v_grid.y = (unsigned int)grid_divide( height, v_block.y );

        dim3 d_block( 32, 1 );
        dim3 d_grid;
        d_grid.x = (unsigned int)grid_divide( width,  d_block.x );
        d_grid.y = (unsigned int)grid_divide( height, d_block.y );

        if(octave==0){
            downscale_by_2<<<h_grid,h_block>>>(base->array,
                                               _octaves[octave  ].getData(0));
        }else{
            downscale_by_2<<<h_grid,h_block>>>(_octaves[octave-1].getData(0),
                                               _octaves[octave  ].getData(0));
        }

    }

    //Performing the gaussing
    for(uint32_t octave=0; octave<_num_octaves; octave++) {
        const int width  = _octaves[octave].getData(0).getWidth();
        const int height = _octaves[octave].getData(0).getHeight();
        dim3 h_block( 64, 2 );
        dim3 h_grid;

        h_grid.x = (unsigned int)grid_divide( width,  h_block.x );
        h_grid.y = (unsigned int)grid_divide( height, h_block.y );

        dim3 v_block( 64, 2 );
        dim3 v_grid;
        v_grid.x = (unsigned int)grid_divide( width,  v_block.x );
        v_grid.y = (unsigned int)grid_divide( height, v_block.y );

        dim3 d_block( 32, 1 );
        dim3 d_grid;
        d_grid.x = (unsigned int)grid_divide( width,  d_block.x );
        d_grid.y = (unsigned int)grid_divide( height, d_block.y );

        //horizontal
        //  input : _data_tex[level-1]
        //  output: getIntermediateData();
        //vertical:
        //  input : _interm_data_tex
        //  output:
        //      lvl0 : getData(level)
        //      lvl>0: getDogSurface()
        for (uint32_t level = 0; level < _levels; level++) {
            if(level == 0){
                filter_gauss_horiz_v11 <<<h_grid,h_block>>> (
                    _octaves[octave]._data_tex[level],
                    _octaves[octave].getIntermediateData() );
                filter_gauss_horiz_v11 <<<h_grid,h_block>>> (
                    _octaves[octave]._data_tex[level],
                    _octaves[octave].getIntermediateData() );
                filter_gauss_horiz_v11 <<<h_grid,h_block>>> (
                    _octaves[octave]._data_tex[level],
                    _octaves[octave].getIntermediateData() );
            }


            if( level == 0 ) {
                if( octave == 0 ) {
                    dim3 block(V11_EDGE_LEN,V11_EDGE_LEN);
                    dim3 grid((unsigned int)grid_divide( width,  V11_EDGE_LEN ),
                              (unsigned int)grid_divide( height, V11_EDGE_LEN ));

                    filter_gauss_horiz_v11 <<<grid,block>>> (
                        _octaves[octave]._data_tex[level-1],
                        _octaves[octave].getIntermediateData() );
                } else {
                    filter_gauss_horiz_v11 <<<h_grid,h_block>>> (
                        _octaves[octave-1]._data_tex[ _levels-3 ],
                        _octaves[octave].getIntermediateData( ) );
                }
            }
            else {
                filter_gauss_horiz_v11 <<<h_grid,h_block>>> (
                        _octaves[octave]._data_tex[ level-1 ],
                                _octaves[octave].getIntermediateData( ) );
            }



            if( level == 0 ) {
                filter_gauss_vert_v11 <<<v_grid,v_block>>> (
                        _octaves[octave]._interm_data_tex,
                                _octaves[octave].getData( level ) );
            }
            else {
                filter_gauss_vert_v11_dog <<<d_grid,d_block>>> (
                        _octaves[octave]._interm_data_tex,
                                _octaves[octave].getData( level ),
                                _octaves[octave]._data_tex[level-1],
                                _octaves[octave].getDogSurface( ),
                                level-1 );
            }
        }
    }

#else

    for( uint32_t octave=0; octave<_num_octaves; octave++ ) {
        for( uint32_t level=0; level<_levels; level++ ) {
#if 0
        cerr << "Configuration for octave " << octave << endl
             << "  Horiz: layer size: "
             << _octaves[octave].getData(level).getWidth() << "x" << _octaves[octave].getData(level).getHeight() << endl
             << "  Vert: layer size: "
             << _octaves[octave].getIntermediateData().getWidth() << "x" << _octaves[octave].getIntermediateData().getHeight() << endl
             << "  grid: "
             << "(" << grid.x << "," << grid.y << "," << grid.z << ")"
             << " block: "
             << "(" << block.x << "," << block.y << "," << block.z << ")" << endl;
#endif

            const int width  = _octaves[octave].getData(0).getWidth();
            const int height = _octaves[octave].getData(0).getHeight();

            dim3 h_block( 64, 2 );
            dim3 h_grid;
            h_grid.x = (unsigned int)grid_divide( width,  h_block.x );
            h_grid.y = (unsigned int)grid_divide( height, h_block.y );

            dim3 v_block( 64, 2 );
            dim3 v_grid;
            v_grid.x = (unsigned int)grid_divide( width,  v_block.x );
            v_grid.y = (unsigned int)grid_divide( height, v_block.y );

            dim3 d_block( 32, 1 );
            dim3 d_grid;
            d_grid.x = (unsigned int)grid_divide( width,  d_block.x );
            d_grid.y = (unsigned int)grid_divide( height, d_block.y );

            if( level == 0 ) {
                if( octave == 0 ) {
                    dim3 block(V11_EDGE_LEN,V11_EDGE_LEN);
                    dim3 grid((unsigned int)grid_divide( width,  V11_EDGE_LEN ),
                              (unsigned int)grid_divide( height, V11_EDGE_LEN ));

                    filter_gauss_horiz_v11 <<<grid,block>>> (
                        base->array,
                        _octaves[octave].getIntermediateData() );
                } else {
                    //this scales down one size?
                    filter_gauss_horiz_v11_by_2 <<<h_grid,h_block>>> (
                        _octaves[octave-1]._data_tex[ _levels-3 ],
                        _octaves[octave].getIntermediateData( ) );
                }
            }
            else {
                filter_gauss_horiz_v11 <<<h_grid,h_block>>> (
                    _octaves[octave]._data_tex[ level-1 ],
                    _octaves[octave].getIntermediateData( ) );
            }

            if( level == 0 ) {
                filter_gauss_vert_v11 <<<v_grid,v_block>>> (
                    _octaves[octave]._interm_data_tex,
                    _octaves[octave].getData( level ) );
            }
            else {
                filter_gauss_vert_v11_dog <<<d_grid,d_block>>> (
                    _octaves[octave]._interm_data_tex,
                    _octaves[octave].getData( level ),
                    _octaves[octave]._data_tex[level-1],
                    _octaves[octave].getDogSurface( ),
                    level-1 );
            }
        }
    }
#endif
    hipDeviceSynchronize( );
    hipError_t err = hipGetLastError();
    POP_CUDA_FATAL_TEST( err, "filter_gauss_horiz_v11 failed: " );
}

} // namespace popart

