#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <vector>
#include <iomanip>
#include <cmath>
#include <assert.h>

#include "SIFT.h"
#include "debug_macros.h"
#include "write_plane_2d.h"

using namespace std;

PopSift::PopSift( int num_octaves,
                  int S_,
                  int upscale_factor,
                  float threshold,
                  float edgeThreshold,
                  float sigma )
    : _octaves( num_octaves )
    , _scales( max(2,S_) ) // min is 2, GPU restriction */
    , up( upscale_factor )
    , _sigma( sigma )
    , _threshold( threshold ) // SIFT parameter
    , _edgeLimit( edgeThreshold ) // SIFT parameter
{
}

PopSift::~PopSift()
{ }

#define TRY_IMAGE_TWICE 0

void PopSift::init( int w, int h )
{
    if (_octaves < 0) {
        _octaves = max(int (floor( logf( (float)min( w, h ) )
                                   / logf( 2.0f ) ) - 3 + up), 1);
    }

    _upscaled_width  = w << up;
    _upscaled_height = h << up;

    _hst_input_image.allocHost( w, h, popart::CudaAllocated );
    _dev_input_image.allocDev( w, h );

    _initTime    = new popart::KeepTime( 0 );
    _uploadTime  = new popart::KeepTime( 0 );
    _pyramidTime = new popart::KeepTime( 0 );
    _extremaTime = new popart::KeepTime( 0 );

    float sigma = 1.0;

    _initTime->start();

    popart::Pyramid::init_filter( sigma, _scales );
    popart::Pyramid::init_sigma(  sigma, _scales );

    _initTime->stop();

    _baseImg = new popart::Image( _upscaled_width, _upscaled_height );
    _pyramid = new popart::Pyramid( _baseImg, _octaves, _scales );

    /* initializing texture for upscale V5
     */

    memset( &_texDesc, 0, sizeof(hipTextureDesc) );
    _texDesc.normalizedCoords = 1; // address 0..1 instead of 0..width/height
    _texDesc.addressMode[0]   = hipAddressModeClamp;
    _texDesc.addressMode[1]   = hipAddressModeClamp;
    _texDesc.addressMode[2]   = hipAddressModeClamp;
    _texDesc.readMode         = hipReadModeNormalizedFloat; // automatic conversion from uchar to float
    _texDesc.filterMode       = hipFilterModeLinear; // bilinear interpolation

    memset( &_resDesc, 0, sizeof(hipResourceDesc) );
    _resDesc.resType                  = hipResourceTypePitch2D;
    _resDesc.res.pitch2D.devPtr       = _dev_input_image.data;
    _resDesc.res.pitch2D.desc.f       = hipChannelFormatKindUnsigned;
    _resDesc.res.pitch2D.desc.x       = 8;
    _resDesc.res.pitch2D.desc.y       = 0;
    _resDesc.res.pitch2D.desc.z       = 0;
    _resDesc.res.pitch2D.desc.w       = 0;
    assert( _dev_input_image.elemSize() == 1 );
    _resDesc.res.pitch2D.pitchInBytes = _dev_input_image.step;
    _resDesc.res.pitch2D.width        = _dev_input_image.getCols();
    _resDesc.res.pitch2D.height       = _dev_input_image.getRows();

    hipError_t err;
    err = hipCreateTextureObject( &_texture, &_resDesc, &_texDesc, 0 );
    POP_CUDA_FATAL_TEST( err, "Could not create texture object: " );
}

void PopSift::uninit( )
{
    hipError_t err;
    err = hipDestroyTextureObject( _texture );
    POP_CUDA_FATAL_TEST( err, "Could not destroy texture object: " );

    _hst_input_image.freeHost( popart::CudaAllocated );
    _dev_input_image.freeDev( );

    _initTime   ->report( "Time to initialize:                        " );
    _uploadTime ->report( "Time to upload:                            " );
    _pyramidTime->report( "Time to build pyramid:                     " );
    _extremaTime->report( "Time to find extrema & compute orientaton: " );

    delete _initTime;
    delete _uploadTime;
    delete _pyramidTime;
    delete _extremaTime;

    delete _baseImg;
    delete _pyramid;
}

void PopSift::execute( imgStream inp )
{
    assert( inp.data_g == 0 );
    assert( inp.data_b == 0 );

    _uploadTime->start();
    memcpy( _hst_input_image.data, inp.data_r, inp.width * inp.height );
    _hst_input_image.memcpyToDevice( _dev_input_image );
    _baseImg->upscale( _dev_input_image, _texture, 1<<up );
    _uploadTime->stop();

    _pyramidTime->start();
    _pyramid->build( _baseImg );
    _pyramidTime->stop();

    _extremaTime->start();
    _pyramid->find_extrema( _edgeLimit, _threshold );
    _extremaTime->stop();

}

void PopSift::log_to_file()
{
  popart::write_plane2D( "upscaled-input-image.pgm",
                         true, // is stored on device
                         _baseImg->array );

  for( int o=0; o<_octaves; o++ ) {
      for( int s=0; s<_scales+3; s++ ) {
          _pyramid->download_and_save_array( "pyramid", o, s );
      }
  }
  for( int o=0; o<_octaves; o++ ) {
      _pyramid->download_and_save_descriptors( "pyramid", o );
  }
}



